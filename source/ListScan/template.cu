#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <wb.h>

#define BLOCK_SIZE 512 //do not change this

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

__global__ void scan(float *input, float *output, float *aux, int len) {
    //@@ Modify the body of this kernel to generate the scanned blocks
    //@@ Make sure to use the workefficient version of the parallel scan
    //@@ Also make sure to store the block sum to the aux array 
     
}

__global__ void addScannedBlockSums(float *output, float *aux, int len) {
	//@@ Modify the body of this kernel to add scanned block sums to 
	//@@ all values of the scanned blocks

}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output 1D list
  float *deviceInput;
  float *deviceOutput;
  float *deviceAuxArray, *deviceAuxScannedArray;
  int numElements; // number of elements in the input/output list. 
				   
  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &numElements);
  hostOutput = (float *)malloc(numElements * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The number of input elements in the input is ",
        numElements);

  wbTime_start(GPU, "Allocating device memory.");
  //@@ Allocate device memory
  //you can assume that deviceAuxArray size would not need to be more than BLOCK_SIZE*2 (i.e., 1024)
  wbTime_stop(GPU, "Allocating device memory.");

  wbTime_start(GPU, "Clearing output device memory.");
  //@@ zero out the deviceOutput using hipMemset() by uncommenting the below line
  //wbCheck(hipMemset(deviceOutput, 0, numElements * sizeof(float)));
  wbTime_stop(GPU, "Clearing output device memory.");

  wbTime_start(GPU, "Copying input host memory to device.");
  //@@ Copy input host memory to device	
  wbTime_stop(GPU, "Copying input host memory to device.");

  //@@ Initialize the grid and block dimensions here

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Modify this to complete the functionality of the scan
  //@@ on the deivce
  //@@ You need to launch scan kernel twice: 1) for generating scanned blocks 
  //@@ (hint: pass deviceAuxArray to the aux parameter)
  //@@ and 2) for generating scanned aux array that has the scanned block sums. 
  //@@ (hint: pass NULL to the aux parameter)
  //@@ Then you should call addScannedBlockSums kernel.
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output device memory to host");
  //@@ Copy results from device to host	
  wbTime_stop(Copy, "Copying output device memory to host");

  wbTime_start(GPU, "Freeing device memory");
  //@@ Deallocate device memory
  wbTime_stop(GPU, "Freeing device memory");

  wbSolution(args, hostOutput, numElements);

  free(hostInput);
  free(hostOutput);

  return 0;
}
